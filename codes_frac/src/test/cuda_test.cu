#include "hip/hip_runtime.h"
#include <iostream> 
#include <stdlib.h> 
#include <time.h>
#include "complex.h"
#include "whu_constants_and_definitions.h"
#include "TComplex.h"
using namespace std;
__device__ complex cro(
                       double sx,
                       double am1,
                       double am2) 
{
  double t1=(am1+am2) * (am1 + am2); // double t1=pow((am1+am2),2);
  double t2=(am1-am2) * (am1 - am2); // double t2=pow((am1-am2),2);
  double st=(sx-t1)*(sx-t2);
  double cro=sqrt(fabs(st))/sx;
  complex result;
  if (st<0.) result.y=cro;
  else result.x=cro;
  return  result;
}

__global__ void propogator980(
                                 double mass,
                                 double g11,
                                 double g22,
                                 double sx,
                                 double *test_r,
                                 double *test_i)
{
  complex ci(0,1);
  double rm=mass*mass;
  complex propogator980=1.0/(rm-sx-ci*(g11*cro(sx,rp,rp)+g22*cro(sx,rk,rk)));
  test_r= propogator980.x;
  test_i= propogator980.y;
}
/*__global__ void propogator2(
                            double mass,
                            double g11,
                            double g22,
                            double *sx,
                            double *b2qjvf2,
                            double *wu,
                            double *w0p22,
                            double *fCF_real,
                            double *fCF_imag,
                            int numbers)
{
  int id = threadIdx.x+blockDim.x*blockIdx.x;
  if(id<numbers){
    id=threadIdx.x+blockDim.x*blockIdx.x;
    complex crp1 = propogator980(mass, g11, g22, sx[id]);
    complex cr0p11 = crp1 / b2qjvf2[id];
    complex result;
    //01 contribution
    result = wu[id] * crp1;
    fCF_real[id] =result.x; 
    fCF_imag[id] =result.y; 
    result = wu[id+numbers] * crp1;
    fCF_real[id+numbers] =result.x; 
    fCF_imag[id+numbers] =result.y; 

    //02 contribution
    result = w0p22[id] * cr0p11;
    fCF_real[id+numbers*2] =result.x; 
    fCF_imag[id+numbers*2] =result.y; 
    result = w0p22[id+numbers] * cr0p11;
    fCF_real[id+numbers*3] =result.x; 
    fCF_imag[id+numbers*3] =result.y;
  }

}
TComplex cpu_cro(double sx, double am1, double am2)  {
    TComplex ci(0, 1);
    double t1 = (am1 + am2) * (am1 + am2);  // double t1=pow((am1+am2),2);
    double t2 = (am1 - am2) * (am1 - am2);  // double t2=pow((am1-am2),2);
    double st = (sx - t1) * (sx - t2);
    double cro = sqrt(fabs(st)) / sx;
    TComplex result = cro;
    if (st < 0.) result = cro * ci;
    return result;
}
*/
TComplex cpu_propogator980(double mass, double g11, double g22,
                                    double sx)  {
    TComplex ci(0, 1);
    double rm = mass * mass;
    TComplex propogator980 =
        1.0 / (rm - sx - ci * (g11 * cpu_cro(sx, rp, rp) + g22 * cpu_cro(sx, rk, rk)));
    return propogator980;
}
void cpu_propogator2(double mass, double g11, double g22,
                                  double *sx, double *b2qjvf2, double *wu,
                                  double *w0p22, TComplex *fCF0, TComplex *fCF1,
                                  int vec_size) {
    for (int i = 0; i < vec_size; i++) {
        TComplex crp1 = cpu_propogator980(mass, g11, g22, sx[i]);
        TComplex cr0p11 = crp1 / b2qjvf2[i];

        // 01 contribution
        fCF0[i + vec_size * 0] = wu[i + vec_size * 0] * crp1;
        fCF0[i + vec_size * 1] = wu[i + vec_size * 1] * crp1;

        // 02 contribution
        fCF1[i + vec_size * 0] = w0p22[i + vec_size * 0] * cr0p11;
        fCF1[i + vec_size * 1] = w0p22[i + vec_size * 1] * cr0p11;
    }
}

int main()
{
  int numbers = 1000;
  double *pa,*pb,*pc,*pd;
  double *pda,*pdb,*pdc,*pdd,*pde,*pdf,*pfr,*pfi;
  TComplex *pe;
  pa = new double[numbers];
  pb = new double[numbers];
  pc = new double[numbers*2];
  pd = new double[numbers*2];
  pfr = new double[numbers*4];
  pfi = new double[numbers*4];

  pe = new TComplex[numbers*4];
  for(int i=0;i<numbers;i++)
    {
      pa[i]=rand()/(double)RAND_MAX;
      pb[i]=rand()/(double)RAND_MAX;
      pc[i]=rand()/(double)RAND_MAX;
      pd[i]=rand()/(double)RAND_MAX;
      pc[i+numbers]=rand()/(double)RAND_MAX;
      pd[i+numbers]=rand()/(double)RAND_MAX;
    }

  hipMalloc((void**)&pda,numbers*sizeof(double));
  hipMalloc((void**)&pdb,numbers*sizeof(double));
  hipMalloc((void**)&pdc,2*numbers*sizeof(double));
  hipMalloc((void**)&pdd,2*numbers*sizeof(double));
  hipMalloc((void**)&pde,4*numbers*sizeof(double));
  hipMalloc((void**)&pdf,4*numbers*sizeof(double));

  hipMemcpyAsync(pda,pa,numbers*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpyAsync(pdb,pb,numbers*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpyAsync(pdc,pc,2*numbers*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpyAsync(pdd,pd,2*numbers*sizeof(double),hipMemcpyHostToDevice);

  propogator2<<<(numbers+63)/64,64>>>(1,1,1,pda,pdb,pdc,pdd,pde,pdf,numbers);
  hipMemcpyAsync(pfr,pde,4*numbers*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpyAsync(pfi,pdf,4*numbers*sizeof(double),hipMemcpyDeviceToHost);
  cpu_propogator980(1,1,1,pa,pb,pc,pd,pe,,numbers);
  for(int i=0;i<20;i++)
    {
      cout<<"cpu p2 "<<" : "<<pe[0]<<"  gpu :"<<pfr[0]<<"+ "<<pfi[0]<<endl;
    }
   return 0;
}


