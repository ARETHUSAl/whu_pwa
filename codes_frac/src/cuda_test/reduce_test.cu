#include<iostream>
#include <unistd.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include <sys/time.h>
const int threads_per_block=64;
clock_t start,end;
__global__ void reduce(int *arrays,int numbers,int *result)
{
  int id = threadIdx.x + blockDim.x*blockIdx.x;
  __shared__ int s_arrays[threads_per_block];
  if (id< numbers)
  {
    s_arrays[threadIdx.x]=arrays[id];
  }
  else
  {
    s_arrays[threadIdx.x]=0;
  }
      __syncthreads();
  for(int i = blockDim.x/2;i>=1;i/=2)
    {
      if(threadIdx.x<i)
      {
      s_arrays[threadIdx.x]+=s_arrays[threadIdx.x+i];
      }
      __syncthreads();
    }
  result[blockIdx.x]=s_arrays[0];
}
int main(int argc, char *argv[])
{
  int numbers=atoi(argv[1]);
  int *a = new int[numbers];
  for(int i=0;i<numbers;i++)
    a[i]=1;
  int *b = new int[numbers];
  int *c = new int[numbers];
  int *da,*db;
  hipMalloc((void**)&da,numbers*sizeof(int));
  hipMalloc((void**)&db,numbers*sizeof(int));
  hipMemcpyAsync(da,a,numbers*sizeof(int),hipMemcpyHostToDevice);
  int count = numbers;
  while(1)
    {
      reduce<<<(count+63)/64,threads_per_block>>>(da,count,db);
      hipDeviceSynchronize();
      count=(count+63)/64;
      std::cout<<count<<std::endl;
      
      if(count==1)
        {
          hipMemcpyAsync(b,db,numbers*sizeof(int),hipMemcpyDeviceToHost);
          break;
        }
      reduce<<<(count+63)/64,threads_per_block>>>(db,count,da);
      hipDeviceSynchronize();
      std::cout<<count<<std::endl;
      count=(count+63)/64;
      if(count==1)
        {
          hipMemcpyAsync(b,da,numbers*sizeof(int),hipMemcpyDeviceToHost);
          break;
        }

    }

  for (int i=0;i<numbers;i++)
    {
      c[0]+=a[i];
    }
  start=clock();
  std::cout<<"b0 "<<b[0]<<" c0 "<<c[0]<<std::endl;
  return 0;
}
